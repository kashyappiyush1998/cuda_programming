#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <chrono>
#include <iostream>
#include <cstdlib>
#include <vector>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>


__global__ void convertColorToGrayScaleKernel(float* A, float* B, int m, int n){
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if(col < m && row < n){
        int grayOffset = row*m + col;
        float r = A[grayOffset * 3 + 0];
        float g = A[grayOffset * 3 + 1];
        float b = A[grayOffset * 3 + 2];

        B[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
    }
}

void convertColorToGrayScale(float* h_A, float* h_B, int m, int n){
    auto start_assigning = std::chrono::high_resolution_clock::now();
    
    int size_d_A = 3 * sizeof(float) * m * n;
    int size_d_B = sizeof(float) * m * n;
    float *d_A, *d_B;
    
    hipMalloc((void **)&d_A, size_d_A);
    hipMemcpy(d_A, h_A, size_d_A, hipMemcpyHostToDevice);
    hipMalloc((void **)&d_B, size_d_B);

    auto stop_assigning = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop_assigning - start_assigning);
    std::cout << "Cuda Assigning time taken : " << duration.count() << std::endl;

    auto start_cuda_execution = std::chrono::high_resolution_clock::now();
    dim3 nGrid(ceil(m/16), ceil(n/16), 1);
    dim3 nBlocks(16, 16, 1);
    convertColorToGrayScaleKernel<<<nGrid, nBlocks>>> (d_A, d_B, m, n);

    auto stop_cuda_execution = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop_cuda_execution - start_cuda_execution);

    std::cout << "Cuda Execution time taken  on cuda : " << duration.count() << std::endl;
    
    hipMemcpy(h_B, d_B, size_d_B, hipMemcpyDeviceToHost);

    hipFree(d_A); hipFree(d_B);
}

int main(){
    std::cout << "Parallel Run" << std::endl;
    auto start_assigning = std::chrono::high_resolution_clock::now();
    
    std::string inpFilePath = "/data/rishubh/piyush/cuda_programming/images/bike_500_11zon.png";
    cv::Mat img = cv::imread(inpFilePath, cv::IMREAD_COLOR);
    cv::Mat img2;
    if(img.empty())
    {
        std::cout << "Could not read the image: " << inpFilePath << std::endl;
        return 1;
    }
    cv::resize(img, img, cv::Size(1024, 1024));
    img.convertTo(img, CV_32FC3);

    std::cout << img.rows << ", " << img.cols << std::endl;
    // std::string outFilePath = "/data/rishubh/piyush/cuda_programming/output_images/bike_500_asitis.png";
    // cv::imwrite(outFilePath, img);

    int m = 1024 , n = 1024;

    int length = m*n;
    float* h_A = (float*)malloc(3 * length * sizeof(float));
    float* h_B = (float*)malloc(length * sizeof(float));

    std::cout << "Memory allocated"<< std::endl;
    std::cout << 3 * length << " " << length << std::endl;
    std::cout<< sizeof(float)<<std::endl;
// 3145728, 1048576
    int offset = 0;
    // cv::Vec3f channels = img.at<cv::Vec3f>(1023,1023);

    for(int i=0; i<m; i++){
        for(int j=0; j<n; j++){
            offset = (i*m+j)*3;
            // std::cout << i << " " << j << " " << offset<<std::endl;
            cv::Vec3f channels = img.at<cv::Vec3f>(i,j);
            h_A[offset + 0] = channels[0];//std::rand()%255;
            h_A[offset + 1] = channels[1];//std::rand()%255;
            h_A[offset + 2] = channels[2];//std::rand()%255;

            h_B[i*m+j] = 0.0f; 
        }
    }
    std::cout << "Memory filled with values" << std::endl;

    auto stop_assigning = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop_assigning - start_assigning);

    std::cout << "Assigning time taken : " << duration.count() << std::endl;
    
    auto start_cuda_execution = std::chrono::high_resolution_clock::now();

    convertColorToGrayScale(&h_A[0], &h_B[0], m, n);
    
    auto stop_cuda_execution = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop_cuda_execution - start_cuda_execution);
    
    cv::Mat out = cv::Mat::zeros(cv::Size(img.cols, img.rows), CV_32FC1);

    for(int i=0; i<m; i++){
        for(int j=0; j<n; j++){
            offset = (i*m+j);
            out.at<float>(i,j) = h_B[offset];
        }
    }

    std::string outFilePath = "/data/rishubh/piyush/cuda_programming/output_images/bike_500_grayscale.png";
    cv::imwrite(outFilePath, out);

    for(int i=0; i<3; i++){
        for(int j=0; j<3; j++){
            std::cout << h_B[j + i] << " , ";
        }
        std::cout<< std::endl;
    }
    std::cout<< std::endl;

    std::cout << "Parallel Run Completed" << std::endl;
    
}