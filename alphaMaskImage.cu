#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <chrono>
#include <iostream>
#include <cstdlib>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

__device__ void blurImageChannelKernel() {

}

__global__ void alphaMaskImageKernel(float* a, float* b, int m, int n){
    int row = blockDim.x * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if(row < m && col < n){
        int offset = (row * m + col);
        float mask_0to1 = b[offset]/255.0f;
        a[3*offset] *= mask_0to1;
        a[3*offset + 1] *= mask_0to1;
        a[3*offset + 2] *= mask_0to1;
    }
}

void alphaMaskImage(float* h_A, float* h_B, int m, int n){
    auto start_assigning = std::chrono::high_resolution_clock::now();
    
    int size_d_A = 3 * sizeof(float) * m * n;
    int size_d_B = sizeof(float) * m * n;
    float *d_A, *d_B;
    
    hipMalloc((void **)&d_A, size_d_A);
    hipMemcpy(d_A, h_A, size_d_A, hipMemcpyHostToDevice);
    hipMalloc((void **)&d_B, size_d_B);
    hipMemcpy(d_B, h_B, size_d_B, hipMemcpyHostToDevice);

    auto stop_assigning = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop_assigning - start_assigning);
    std::cout << "Cuda Assigning time taken : " << duration.count() << std::endl;

    auto start_cuda_execution = std::chrono::high_resolution_clock::now();
    dim3 nGrid(ceil(m/16), ceil(n/16), 1);
    dim3 nBlocks(16, 16, 1);
    alphaMaskImageKernel<<<nGrid, nBlocks>>> (d_A, d_B, m, n);

    auto stop_cuda_execution = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop_cuda_execution - start_cuda_execution);

    std::cout << "Cuda Execution time taken  on cuda : " << duration.count() << std::endl;
    
    hipMemcpy(h_A, d_A, size_d_A, hipMemcpyDeviceToHost);

    hipFree(d_A); hipFree(d_B);
}

cv::Mat loadResizeImage(std::string path){
    cv::Mat img = cv::imread(path, cv::IMREAD_COLOR);
    if(img.empty())
    {
        std::cout << "Could not read the image: " << path << std::endl;
        return img;
    }
    cv::resize(img, img, cv::Size(1024, 1024));
    img.convertTo(img, CV_32FC3);

    std::cout << img.rows << ", " << img.cols << std::endl;
    return img;
}

int main(){
    std::cout << "Parallel Run" << std::endl;
    auto start_assigning = std::chrono::high_resolution_clock::now();

    std::string inpFilePath = "/data/rishubh/piyush/cuda_programming/images/bracelet.jpg";
    std::string maskFilePath = "/data/rishubh/piyush/cuda_programming/images/bracelet_alpha.png";
    
    cv::Mat img = loadResizeImage(inpFilePath);
    cv::Mat mask = loadResizeImage(maskFilePath);
    // cv::cvtColor(mask, mask, cv::CV_BGR2BGRA);
    cv::cvtColor(mask, mask, cv::COLOR_BGR2GRAY);

    int m = 1024 , n = 1024;
    int length = m*n;
    float* h_A = (float*)malloc(3 * length * sizeof(float));
    float* h_B = (float*)malloc(length * sizeof(float));

    std::cout << "Memory allocated"<< std::endl;
    int offset = 0;

    for(int i=0; i<m; i++){
        for(int j=0; j<n; j++){
            offset = (i*m+j);
            // std::cout << i << " " << j << " " << offset<<std::endl;
            cv::Vec3f channels = img.at<cv::Vec3f>(i,j);
            h_A[3*offset + 0] = channels[0];  h_B[offset + 0] = mask.at<float>(i, j);
            h_A[3*offset + 1] = channels[1];  
            h_A[3*offset + 2] = channels[2];  

        }
    }
    std::cout << "Memory filled with values" << std::endl;

    auto stop_assigning = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop_assigning - start_assigning);

    std::cout << "Assigning time taken : " << duration.count() << std::endl;
    
    auto start_cuda_execution = std::chrono::high_resolution_clock::now();

    alphaMaskImage(&h_A[0], &h_B[0], m, n);
    
    auto stop_cuda_execution = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop_cuda_execution - start_cuda_execution);
    
    cv::Mat out = cv::Mat::zeros(cv::Size(img.cols, img.rows), CV_32FC3);

    cv::Vec3f assignPixels;
    for(int i=0; i<m; i++){
        for(int j=0; j<n; j++){
            offset = 3*(i*m+j);
            assignPixels[0] = h_A[offset + 0];// * h_B[offset/3] / 255.0;
            assignPixels[1] = h_A[offset + 1];// * h_B[offset/3] / 255.0;;
            assignPixels[2] = h_A[offset + 2];// * h_B[offset/3] / 255.0;;
            out.at<cv::Vec3f>(i,j) = assignPixels;
        }
    }

    std::string outFilePath = "/data/rishubh/piyush/cuda_programming/output_images/braclet_transparent.png";
    cv::imwrite(outFilePath, out);

    for(int i=0; i<3; i++){
        for(int j=0; j<3; j++){
            std::cout << h_A[j + i] << " , ";
        }
        std::cout<< std::endl;
    }
    std::cout<< std::endl;

    std::cout << "Parallel Run Completed" << std::endl;
    
}